#include "hip/hip_runtime.h"

// @Date    : 2019-06-24 09:52:56
// @Author  : Mengji Zhang (zmj_xy@sjtu.edu.cn)

#include "hip/hip_runtime.h"
#include "test.h"
#include "cpu_bitmap.h"

#define DIM 1024
#define PI 3.1415926535897932f

__global__ void kernel(unsigned char *ptr){
	int x=threadIdx.x+blockIdx.x*blockDim.x;
	int y=threadIdx.y+blockIdx.y*blockDim.y;
	int offset=x+y*blockDim.x*gridDim.x;

	__shared__ float shared[16][16];
	const float period=128.f;
	shared[threadIdx.x][threadIdx.y]=255*(sinf(x*2.f*PI/period)+1.f)*(sinf(y*2.f*PI/period)+1.f)/4.f;
	__syncthreads();

	ptr[offset*4+0]=0;
	ptr[offset*4+1]=shared[15-threadIdx.x][15-threadIdx.y];
	ptr[offset*4+2]=0;
	ptr[offset*4+3]=255;

}
int main(){
	CPUBitmap bitmap(DIM,DIM);
	unsigned char *dev_bitmap;

	hipMalloc((void**)&dev_bitmap,bitmap.image_size());

	dim3 grids(DIM/16,DIM/16);
	dim3 threads(16,16);

	kernel<<<grids,threads>>>(dev_bitmap);

	hipMemcpy(bitmap.get_ptr(),dev_bitmap,bitmap.image_size(),hipMemcpyDeviceToHost);

	bitmap.display_and_exit();
	hipFree(dev_bitmap);
}