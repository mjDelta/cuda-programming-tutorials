#include "hip/hip_runtime.h"


// @Date    : 2019-06-23 09:55:44
// @Author  : Mengji Zhang (zmj_xy@sjtu.edu.cn)

#include "../headers/book.h"
#define N 10
__global__ void add(int *a,int *b,int *c){
	int tid=threadIdx.x;
	if (tid<N){
		c[tid]=a[tid]+b[tid];
	}
}
int main(){
	int a[N],b[N],c[N];
	int *dev_a,*dev_b,*dev_c;

	hipMalloc((void**)&dev_a,N*sizeof(int));
	hipMalloc((void**)&dev_b,N*sizeof(int));
	hipMalloc((void**)&dev_c,N*sizeof(int));

	for(int i=0;i<N;i++){
		a[i]=i;
		b[i]=i*i;
	}
	hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,N*sizeof(int),hipMemcpyHostToDevice);

	add<<<1,N>>>(dev_a,dev_b,dev_c);

	hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);

	for(int i=0;i<N;i++){
		printf("%d+%d=%d\n",a[i],b[i],c[i]);
	}
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	getchar();

	return 0;
}