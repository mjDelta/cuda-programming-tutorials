#include "hip/hip_runtime.h"

// @Date    : 2019-06-25 09:41:33
// @Author  : Mengji Zhang (zmj_xy@sjtu.edu.cn)

#include "hip/hip_runtime.h"
#include "book.h"
#include "../common/cpu_bitmap.h"

#define INF 2e10f
struct Sphere{
	float r,g,b;
	float radius;
	float x,y,z;

	__device__ float hit(float ox,float oy,float *n){
		float dx=ox-x;
		float dy=oy-y;
		if (dx*dx+dy*dy<radius*radius){
			float dz=sqrtf(radius*radius-dx*dx-dy*dy);
			*n=dz/float(radius);
			return dz+z;
		}
		return -INF;
	}
};

#define rnd(x) (x*rand()/float(RAND_MAX))
#define SPHERES 20
#define DIM 1024

int main(){
	for(int i=0;i<10;i++)
		printf("%f\n",rnd(255));
	CPUBitmap bitmap(DIM,DIM);
	// bitmap.display_and_exit();
	getchar();
	return 0;
}