#include "hip/hip_runtime.h"

// @Date    : 2019-06-25 09:41:33
// @Author  : Mengji Zhang (zmj_xy@sjtu.edu.cn)

#include "../headers/hip/hip_runtime.h"
#include "../headers/book.h"
#include "../headers/bitmap.h"

#define INF 2e10f
struct Sphere{
	float r,g,b;
	float radius;
	float x,y,z;

	__device__ float hit(float ox,float oy,float *n){
		float dx=ox-x;
		float dy=oy-y;
		if (dx*dx+dy*dy<radius*radius){
			float dz=sqrtf(radius*radius-dx*dx-dy*dy);
			*n=dz/float(radius);
			return dz+z;
		}
		return -INF;
	}
};

#define rnd(x) (x*rand()/float(RAND_MAX))
#define SPHERES 40
#define DIM 1024

__constant__ Sphere s[SPHERES];

__global__ void kernel(char *ptr){
	int x=threadIdx.x+blockIdx.x*blockDim.x;
	int y=threadIdx.y+blockIdx.y*blockDim.y;
	int offset=x+y*blockDim.x*gridDim.x;

	float ox=(x-DIM/2);
	float oy=(y-DIM/2);

	float r=0,g=0,b=0;
	float maxz=-INF;
	for(int i=0;i<SPHERES;i++){
		float n;
		float t=s[i].hit(ox,oy,&n);
		if (t>maxz){
			float fscale=n;
			r=s[i].r*fscale;
			g=s[i].g*fscale;
			b=s[i].b*fscale;
			// maxz=t;
		}
	}
	ptr[offset*3+0]=(int)(r*255);
	ptr[offset*3+1]=(int)(g*255);
	ptr[offset*3+2]=(int)(b*255);
}
int main(){
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	CPUBitmap bitmap(DIM,DIM);
	char *savePath="test07.bmp";
	char *dev_bitmap;
	hipMalloc(&dev_bitmap,bitmap.size);

	Sphere *temp_s=(Sphere*)malloc(sizeof(Sphere)*SPHERES);
	for(int i=0;i<SPHERES;i++){
		temp_s[i].r=rnd(1.f);
		temp_s[i].g=rnd(1.f);
		temp_s[i].b=rnd(1.f);
		temp_s[i].x=rnd(DIM)-DIM/2;
		temp_s[i].y=rnd(DIM)-DIM/2;
		temp_s[i].z=rnd(DIM)-DIM/2;
		temp_s[i].radius=rnd(100.f)+50;
	}
	hipMemcpyToSymbol(HIP_SYMBOL(s),temp_s,sizeof(Sphere)*SPHERES);
	free(temp_s);

	dim3 grids(DIM/16,DIM/16);
	dim3 threads(16,16);

	kernel<<<grids,threads>>>(dev_bitmap);

	hipMemcpy(bitmap.pixels,dev_bitmap,bitmap.size,hipMemcpyDeviceToHost);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("Time cost:%f",elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	bitmap.saveBitmap(savePath);
	hipFree(dev_bitmap);

	getchar();
	return 0;
}