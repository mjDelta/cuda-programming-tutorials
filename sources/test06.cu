#include "hip/hip_runtime.h"

// @Date    : 2019-06-24 09:52:56
// @Author  : Mengji Zhang (zmj_xy@sjtu.edu.cn)

#include "../headers/hip/hip_runtime.h"
#include "../headers/book.h"
#include "../headers/bitmap.h"

#define DIM 1024
#define PI 3.1415926535897932f

__global__ void kernel(unsigned char *ptr){
	int x=threadIdx.x+blockIdx.x*blockDim.x;
	int y=threadIdx.y+blockIdx.y*blockDim.y;
	int offset=x+y*blockDim.x*gridDim.x;

	__shared__ float shared[16][16];
	const float period=128.f;
	shared[threadIdx.x][threadIdx.y]=255*(sinf(x*2.f*PI/period)+1.f)*(sinf(y*2.f*PI/period)+1.f)/4.f;
	__syncthreads();

	ptr[offset*3+0]=0;
	ptr[offset*3+1]=shared[15-threadIdx.x][15-threadIdx.y];
	ptr[offset*3+2]=0;

}
int main(){
	CPUBitmap bitmap(DIM,DIM);
	unsigned char *dev_bitmap;
	char *savePath="test06.bmp";

	hipMalloc((void**)&dev_bitmap,bitmap.size);

	dim3 grids(DIM/16,DIM/16);
	dim3 threads(16,16);

	kernel<<<grids,threads>>>(dev_bitmap);

	hipMemcpy(bitmap.pixels,dev_bitmap,bitmap.size,hipMemcpyDeviceToHost);

	bitmap.saveBitmap(savePath);
	hipFree(dev_bitmap);
	getchar();
}