#include "hip/hip_runtime.h"

// @Date    : 2019-05-19 21:26:29
// @Author  : Mengji Zhang (zmj_xy@sjtu.edu.cn)

#include "../headers/book.h"
#define N 10
__global__ void add(int *a,int *b,int *c){
	int tid=blockIdx.x;
	if(tid<N){
		c[tid]=a[tid]+b[tid];
	}
}

int main(void){
	int a[N],b[N],c[N];
	int *dev_a,*dev_b,*dev_c;
	
	hipMalloc((void**)&dev_a,N*sizeof(int));
	hipMalloc((void**)&dev_b,N*sizeof(int));
	hipMalloc((void**)&dev_c,N*sizeof(int));

	//assign values for a and b in CPU
	for(int i=0;i<N;i++){
		a[i]=-i;
		b[i]=i*i;
	}

	//copy a and b to GPU
	hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,N*sizeof(int),hipMemcpyHostToDevice);

	//run on GPU
	add<<<N,1>>>(dev_a,dev_b,dev_c);

	//copy result from GPU to CPU
	hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);

	//result display
	for(int i=0;i<N;i++){
		printf("%d+%d=%d\n",a[i],b[i],c[i]);
	}

	//free cuda memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	printf("The program is done. Enter anything to exit.");
	getchar();
	return 0;
}


