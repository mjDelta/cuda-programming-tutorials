#include "hip/hip_runtime.h"

// @Date    : 2019-06-23 14:50:31
// @Author  : Mengji Zhang (zmj_xy@sjtu.edu.cn)

#include "../headers/book.h"

#define imin(a,b) (a<b?a:b)

const int N=1024;
const int threadsPerBlock=256;
const int blocksPerGrid=imin(32,(N+threadsPerBlock-1)/threadsPerBlock);

__global__ void dot(float *a, float *b,float *c){
	__shared__ float cache[threadsPerBlock];
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
	int cacheIdx=threadIdx.x;

	float temp=0.;
	while(tid<N){
		temp+=a[tid]*b[tid];
		tid+=blockDim.x*gridDim.x;
	}
	cache[cacheIdx]=temp;
	__syncthreads();

	int i=blockDim.x/2;
	while(i!=0){
		if (cacheIdx<i){
			cache[cacheIdx]+=cache[cacheIdx+i];
		}
		__syncthreads();
		i/=2;
	}
	if (cacheIdx==0){
		c[blockIdx.x]=cache[0];
	}
}

int main(){
	float *a,*b,c,*partial_c;
	float *dev_a,*dev_b,*dev_c;
	c=0;

	a=(float*)malloc(N*sizeof(float));
	b=(float*)malloc(N*sizeof(float));
	partial_c=(float*)malloc(blocksPerGrid*sizeof(float));

	hipMalloc((void**)&dev_a,N*sizeof(float));
	hipMalloc((void**)&dev_b,N*sizeof(float));
	hipMalloc((void**)&dev_c,blocksPerGrid*sizeof(float));

	for(int i=0;i<N;i++){
		a[i]=i;
		b[i]=2*i;
	}

	hipMemcpy(dev_a,a,N*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,N*sizeof(float),hipMemcpyHostToDevice);


	dot<<<blocksPerGrid,threadsPerBlock>>>(dev_a,dev_b,dev_c);

	hipMemcpy(partial_c,dev_c,blocksPerGrid*sizeof(float),hipMemcpyDeviceToHost);

	#define sum_square(x) (x*(x+1)*(2*x+1)/6)

	for(int i=0;i<blocksPerGrid;i++){
		c+=partial_c[i];
	}


	printf("Does %f==%f?\n",c,2*sum_square((float)(N-1)));

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	free(a);
	free(b);
	free(partial_c);
	getchar();



}