#include "hip/hip_runtime.h"


// @Date    : 2019-06-23 10:10:30
// @Author  : Mengji Zhang (zmj_xy@sjtu.edu.cn)

#include "test.h"
#define N (33*1024)

__global__ void add(int *a,int *b,int *c){
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
	while(tid<N){
		c[tid]=a[tid]+b[tid];
		tid+=blockDim.x*gridDim.x;
	}
}

int main(){
	int a[N],b[N],c[N];
	int *dev_a,*dev_b,*dev_c;

	HANDLE_ERROR(hipMalloc((void**)&dev_a,N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b,N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c,N*sizeof(int)));

	for(int i=0;i<N;i++){
		a[i]=i;
		b[i]=i*i;
	}

	HANDLE_ERROR(hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b,b,N*sizeof(int),hipMemcpyHostToDevice));

	add<<<(N+127)/128,128>>>(dev_a,dev_b,dev_c);

	HANDLE_ERROR(hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost));

	bool success=true;
	for(int i=0;i<N;i++){
		if (a[i]+b[i]!=c[i]){
			printf("%d+%d!=%d\n",a[i],b[i],c[i]);
			success=false;
		}
	}

	if (success){printf("It's right!\n");}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	getchar();

	return 0;
}