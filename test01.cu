#include "hip/hip_runtime.h"

// @Date    : 2019-05-19 21:26:29
// @Author  : Mengji Zhang (zmj_xy@sjtu.edu.cn)

#include <iostream>
#include <stdio.h>

#include "test.h"
#define N 10
__global__ void add(int *a,int *b,int *c){
	int tid=blockIdx.x;
	if(tid<N){
		c[tid]=a[tid]+b[tid];
	}
}

int main(void){
	int a[N],b[N],c[N];
	int *dev_a,*dev_b,*dev_c;
	
	HANDLE_ERROR(hipMalloc((void**)&dev_a,N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b,N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c,N*sizeof(int)));

	//assign values for a and b in CPU
	for(int i=0;i<N;i++){
		a[i]=-i;
		b[i]=i*i;
	}

	//copy a and b to GPU
	HANDLE_ERROR(hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b,b,N*sizeof(int),hipMemcpyHostToDevice));

	//run on GPU
	add<<<N,1>>>(dev_a,dev_b,dev_c);

	//copy result from GPU to CPU
	HANDLE_ERROR(hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost));

	//result display
	for(int i=0;i<N;i++){
		printf("%d+%d=%d\n",a[i],b[i],c[i]);
	}

	//free cuda memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	printf("The program is done. Enter anything to exit.");
	getchar();
	return 0;

}


